#include "hip/hip_runtime.h"
#include "fp16.hpp"
#include "fp16.cuh"


__global__ void convert_to_fp16(const int n, float* in, fp16* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = fp32tofp16_gpu(in[index]);
  }
}

__global__ void convert_to_fp16(const int n, double* in, fp16* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp32tofp16_gpu(in[index]);
  }
}
__global__ void convert_to_float(const int n,  fp16* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n,  fp16* in, double* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n, const fp16* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }
}

__global__ void convert_to_float(const int n, const fp16* in, double* out) {
  CUDA_KERNEL_LOOP(index, n) {
   out[index] = fp16tofp32_gpu(in[index]);
  }

}
